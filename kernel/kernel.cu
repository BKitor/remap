#include "hip/hip_runtime.h"
#include "kernel.h"
#include "stdio.h"
#include "math.h"
#include "hip/hip_runtime.h"

#define NUM_STREAMS 5

hipStream_t pStreams[NUM_STREAMS];
int stream_idx = 0;
int initalized = 0;

static inline hipStream_t get_stream() {
  if (!initalized) {
    for (int i=0; i<NUM_STREAMS; i++) {
      hipStreamCreate(&pStreams[i]);
    }
    initalized = 1;
  }
  stream_idx = (stream_idx + 1) % NUM_STREAMS;
  return pStreams[stream_idx];
}

// Calculated A = A + B
__global__ void vecAddImpl(float *a, float *b, int n)
{
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n) {
    float tmp_buf = a[id] + b[id];
    a[id] = tmp_buf;
    b[id] = tmp_buf;
  }
}

extern "C" void vecAdd(float *a, float *b, int n) {
  int Db = n < 1024 ? n : 1024;
  int Dg = ceil((float) n / (float) Db);

  int Ns = n * sizeof(float) < 48 * 1024
         ? n * sizeof(float)
         : 48 * 1024;

  hipStream_t stream = get_stream();
  vecAddImpl<<<Dg, Db, Ns, stream>>>(a, b, n);
  hipStreamSynchronize(stream);
}
